#pragma once
#include "kadd.cu"

// Fonction callable depuis C++ / Python
void hostCuda(float *ptrTab, int n)
{
    float *ptrTabGM;
    size_t sizeOctet = sizeof(float) * n;
    hipMalloc(&ptrTabGM, sizeOctet);
    hipMemcpy(ptrTabGM, ptrTab, sizeOctet, hipMemcpyHostToDevice);

    dim3 dg(68, 1, 1);
    dim3 db(128, 1, 1);
    kadd<<<dg, db>>>(ptrTabGM, n);

    hipMemcpy(ptrTab, ptrTabGM, sizeOctet, hipMemcpyDeviceToHost);
    hipFree(ptrTabGM);
}

#include "hip/hip_runtime.h"
#include "activation.cuh"
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_fp16.h>
#include "util.h"
#include "util.cuh"
#include "compat.cuh"

#define NUM_THREADS 256
#define ACT_SILU 0
#define ACT_GELU 1
#define ACT_RELU2 2

__device__ inline half2 clamp_half2_to_finite(half2 v)
{
    const half2 max_h2 = __float2half2_rn(65504.0f);
    const half2 min_h2 = __float2half2_rn(-65504.0f);
    return __hmax2(__hmin2(v, max_h2), min_h2);
}

__device__ __forceinline__ half _silu(half x)
{
    half one = __float2half(1.0f);
    half neg_x = __hneg(x);
    half e = hexp(neg_x);
    half sum = __hadd(one, e);
    half r = hrcp(sum);
    half result = __hmul(x, r);
    return result;
}

__device__ __forceinline__ half2 _silu(half2 x)
{
    half2 one = __float2half2_rn(1.0f);
    half2 neg_x = __hneg2(x);
    half2 e = h2exp(neg_x);
    half2 sum = __hadd2(one, e);
    half2 r = h2rcp(sum);
    half2 result = __hmul2(x, r);
    return result;
}

__device__ __forceinline__ float _silu(float x)
{
    float e     = __expf(-x);
    float recip = __fdividef(1.0f, 1.0f + e);
    return x * recip;
}

__device__ __forceinline__ half _gelu(half x)
{
    float xf = __half2float(x);
    const float c = 0.797884560803f;  // sqrt(2/Pi)
    float tanh_arg = c * (xf + 0.044715f * xf * xf * xf);
    xf = 0.5f * xf * (1.0 + tanh_opt(tanh_arg));
    return __float2half_rn(xf);
}

__device__ __forceinline__ float _gelu(float x)
{
    const float c = 0.797884560803f;  // sqrt(2/Pi)
    float tanh_arg = c * (x + 0.044715f * x * x * x);
    x = 0.5f * x * (1.0 + tanh_opt(tanh_arg));
    return x;
}

__device__ __forceinline__ half2 _gelu(half2 x)
{
    return __halves2half2(_gelu(__low2half(x)), _gelu(__high2half(x)));
}

__device__ __forceinline__ half _relu2(half x)
{
    float xf = __half2float(x);
    xf = fmaxf(0.0f, xf);
    xf = xf * xf;
    return __float2half_rn(xf);
}

__device__ __forceinline__ float _relu2(float x)
{
    x = fmaxf(0.0f, x);
    x = x * x;
    return x;
}

__device__ __forceinline__ half2 _relu2(half2 x)
{
    return __halves2half2(_relu2(__low2half(x)), _relu2(__high2half(x)));
}

template <int activation_type>
__global__ __launch_bounds__(NUM_THREADS)
void act_mul_kernel_h
(
    const half* __restrict__ x,
    const half* __restrict__ y,
    half* __restrict__ z,
    size_t numel
)
{
    size_t idx = (blockIdx.x * NUM_THREADS + threadIdx.x);
    if (idx >= numel / 2) return;

    half2 x2 = ((const half2*) x)[idx];
    half2 y2 = ((const half2*) y)[idx];

    if constexpr (activation_type == ACT_SILU)
        x2 = _silu(x2);
    else if constexpr (activation_type == ACT_GELU)
        x2 = _gelu(x2);

    ((half2*) z)[idx] = __hmul2(x2, y2);
}

template <int activation_type>
__global__ __launch_bounds__(NUM_THREADS)
void act_mul_kernel_f
(
    const float* __restrict__ x,
    const float* __restrict__ y,
    half* __restrict__ z,
    size_t numel
)
{
    size_t idx = (blockIdx.x * NUM_THREADS + threadIdx.x);
    if (idx >= numel / 2) return;

    float2 x2 = ((const float2*) x)[idx];
    float2 y2 = ((const float2*) y)[idx];

    if constexpr (activation_type == ACT_SILU)
    {
        x2.x = _silu(x2.x);
        x2.y = _silu(x2.y);
    }
    else if constexpr (activation_type == ACT_GELU)
    {
        x2.x = _gelu(x2.x);
        x2.y = _gelu(x2.y);
    }
    else if constexpr (activation_type == ACT_RELU2)
    {
        x2.x = _relu2(x2.x);
        x2.y = _relu2(x2.y);
    }

    x2.x *= y2.x;
    x2.y *= y2.y;
    half2 r = __float22half2_rn(x2);
    r = clamp_half2_to_finite(r);
    ((half2*) z)[idx] = r;
}

// silu(x) * y -> z, in-place if z == x or z == y

void silu_mul
(
    const at::Tensor& x,
    const at::Tensor& y,
    at::Tensor& z
)
{
    const at::cuda::OptionalCUDAGuard device_guard(x.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    bool float_input = x.dtype() == at::kFloat;
    if (float_input)
    {
        TORCH_CHECK_DTYPE(y, kFloat);
    }
    else
    {
        TORCH_CHECK_DTYPE(x, kHalf);
        TORCH_CHECK_DTYPE(y, kHalf);
    }

    TORCH_CHECK_DTYPE(z, kHalf);

    size_t numel = x.numel();
    size_t blocks = CEIL_DIVIDE(numel, 2 * NUM_THREADS);
    if (float_input)
    {
        act_mul_kernel_f<ACT_SILU><<<blocks, NUM_THREADS, 0, stream>>>
        (
            (const float*) x.data_ptr(),
            (const float*) y.data_ptr(),
            (half*) z.data_ptr(),
            numel
        );
    }
    else
    {
        act_mul_kernel_h<ACT_SILU><<<blocks, NUM_THREADS, 0, stream>>>
        (
            (const half*) x.data_ptr(),
            (const half*) y.data_ptr(),
            (half*) z.data_ptr(),
            numel
        );
    }
}

// silu(x) * y -> z, in-place if z == x or z == y

void gelu_mul
(
    const at::Tensor& x,
    const at::Tensor& y,
    at::Tensor& z
)
{
    const at::cuda::OptionalCUDAGuard device_guard(x.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    bool float_input = x.dtype() == at::kFloat;
    if (float_input)
    {
        TORCH_CHECK_DTYPE(y, kFloat);
    }
    else
    {
        TORCH_CHECK_DTYPE(x, kHalf);
        TORCH_CHECK_DTYPE(y, kHalf);
    }

    TORCH_CHECK_DTYPE(z, kHalf);

    size_t numel = x.numel();
    size_t blocks = CEIL_DIVIDE(numel, 2 * NUM_THREADS);
    if (float_input)
    {
        act_mul_kernel_f<ACT_GELU><<<blocks, NUM_THREADS, 0, stream>>>
        (
            (const float*) x.data_ptr(),
            (const float*) y.data_ptr(),
            (half*) z.data_ptr(),
            numel
        );
        cuda_check(hipPeekAtLastError());
    }
    else
    {
        act_mul_kernel_h<ACT_GELU><<<blocks, NUM_THREADS, 0, stream>>>
        (
            (const half*) x.data_ptr(),
            (const half*) y.data_ptr(),
            (half*) z.data_ptr(),
            numel
        );
        cuda_check(hipPeekAtLastError());
    }
}

void relu2_mul
(
    const at::Tensor& x,
    const at::Tensor& y,
    at::Tensor& z
)
{
    const at::cuda::OptionalCUDAGuard device_guard(x.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    bool float_input = x.dtype() == at::kFloat;
    if (float_input)
    {
        TORCH_CHECK_DTYPE(y, kFloat);
    }
    else
    {
        TORCH_CHECK_DTYPE(x, kHalf);
        TORCH_CHECK_DTYPE(y, kHalf);
    }

    TORCH_CHECK_DTYPE(z, kHalf);

    size_t numel = x.numel();
    size_t blocks = CEIL_DIVIDE(numel, 2 * NUM_THREADS);
    if (float_input)
    {
        act_mul_kernel_f<ACT_RELU2><<<blocks, NUM_THREADS, 0, stream>>>
        (
            (const float*) x.data_ptr(),
            (const float*) y.data_ptr(),
            (half*) z.data_ptr(),
            numel
        );
    }
    else
    {
        act_mul_kernel_h<ACT_RELU2><<<blocks, NUM_THREADS, 0, stream>>>
        (
            (const half*) x.data_ptr(),
            (const half*) y.data_ptr(),
            (half*) z.data_ptr(),
            numel
        );
    }
}